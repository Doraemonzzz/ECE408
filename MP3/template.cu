#include "hip/hip_runtime.h"

#include <wb.h>
#include <math.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

int d = 16;

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  const int TILE_WIDTH = 16;
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = bx * TILE_WIDTH + tx;
  int col = by * TILE_WIDTH + ty;
  float v = 0;

  int n = ceil((float)numAColumns / TILE_WIDTH);
  for (int i = 0; i < n; i++) {
    // (row, i * TILE_WIDTH + ty)在范围内
    if (row < numARows && (i * TILE_WIDTH + ty) < numAColumns) {
      subTileA[tx][ty] = A[row * numAColumns + i * TILE_WIDTH + ty];
    } else {
      subTileA[tx][ty] = 0;
    }

    // (i * TILE_WIDTH + tx, col)在范围内
    if ((i * TILE_WIDTH + tx) < numBRows && col < numBColumns) {
      subTileB[tx][ty] = B[(i * TILE_WIDTH + tx) * numBColumns + col];
    } else {
      subTileB[tx][ty] = 0;
    }
    // 完成读取数据
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; k++) {
      v += subTileA[tx][k] * subTileB[k][ty];
    }
    __syncthreads();
  }
  if (row < numCRows && col < numCColumns) {
    C[row * numCColumns + col] = v;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **) &deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **) &deviceC, numCRows * numCColumns * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(1.0 * numCRows / d), ceil(1.0 * numCColumns / d), 1);
  dim3 DimBlock(d, d, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>> (deviceA, deviceB, deviceC, 
                                         numARows, numAColumns,
                                         numBRows, numBColumns, 
                                         numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
